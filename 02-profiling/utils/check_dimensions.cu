#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void check_index(void) {
  // pre defined uint3 data types
  printf(
    "[+] threadIdx: (%d, %d, %d) blockIdx: (%d, %d, %d) blockDim: (%d, %d, %d) gridDim: (%d, %d, %d)\n", 
    threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z
  );
}

int main(int argc, char **argv) {
  // define total data elements
  int elements = 6;
  // define grid block structure: choose a block size then calculate grid size from data and block sizes
  // here we define a 1D block with 3 threads and a 1D grid
    dim3 block(3);  // manually defined dim3 data types
    dim3 grid ((elements + block.x -1) / block.x); // grid size rounded up to multiple of block size

  printf("[x] Checking grid/block indices and dimensions.. \n");
  // check grid and block dimensions from host side
  printf("[+] grid.x: %d, grid.y: %d, grid.z: %d\n", grid.x, grid.y, grid.z);
  printf("[+] block.x: %d, block.y: %d, block.z: %d\n", block.x, block.y, block.z);
  
  // check grid and block dimensions from device side 
  check_index <<<grid, block>>> ();
  //reset device before leave leaving
  hipDeviceReset();
  return 0;
}
/*
[x] Checking grid/block indices and dimensions.. 
[+] grid.x: 2, grid.y: 1, grid.z: 1
[+] block.x: 3, block.y: 1, block.z: 1
[+] threadIdx: (0, 0, 0) blockIdx: (1, 0, 0) blockDim: (3, 1, 1) gridDim: (2, 1, 1)
[+] threadIdx: (1, 0, 0) blockIdx: (1, 0, 0) blockDim: (3, 1, 1) gridDim: (2, 1, 1)
[+] threadIdx: (2, 0, 0) blockIdx: (1, 0, 0) blockDim: (3, 1, 1) gridDim: (2, 1, 1)
[+] threadIdx: (0, 0, 0) blockIdx: (0, 0, 0) blockDim: (3, 1, 1) gridDim: (2, 1, 1)
[+] threadIdx: (1, 0, 0) blockIdx: (0, 0, 0) blockDim: (3, 1, 1) gridDim: (2, 1, 1)
[+] threadIdx: (2, 0, 0) blockIdx: (0, 0, 0) blockDim: (3, 1, 1) gridDim: (2, 1, 1)
*/
