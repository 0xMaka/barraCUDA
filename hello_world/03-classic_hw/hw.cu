
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(void) {
  printf("Hello world, from GPU!\n");
}

int main(void) {
  hello<<<1,5>>>();
	hipError_t errSync = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();

	if (errSync != hipSuccess)
    printf("Sync kernel error!\n%s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error!\n%s\n", hipGetErrorString(errAsync));
  
	return 0;
}


