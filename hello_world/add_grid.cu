
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void add(int n, float *x, float *y) {
  // ## cuda c++ provides keywords to return indices of running threads
  // - threadIdx.x contains index of current thread within its block
  // - blockDim.x contains number of threads in the block
  // ... the following is often called a grid-stride-loop.
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i=index; i<n; i+=stride)
    y[i] = x[i] + y[i];
}

int main(void) {
  int N = 1<<20; //20mill elements
  float *x, *y;	
  // allocate unified memory accessible from gpu or cpu
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host (cpu)
  for (int i=0; i<N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // <<< n of thread blocks, n of threads >>>
  // calc blocks needed to get n number of threads
  // divide n by block size (round up)
  int blockSize = 256;
  int numBlocks = (N + blockSize -1) / blockSize;
  // run kernel on 4096 blocks, 256 threads of the gpu
  add<<<numBlocks,blockSize>>>(N,x,y);

//std::cout << "<<< " << numBlocks << ", " << blockSize << " >>>" << std::endl;
  
  // wait for gpu to finish before accessing cpu
  hipDeviceSynchronize();

  // check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i=0; i<N; i++) 
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // free memory
  hipFree(x);
  hipFree(y);
  return 0;
}
