#include "hip/hip_runtime.h"
#include <stdio.h> 
__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(i < n) // if in bounds
    y[i] = (a * x[i]) + y[i]; // single-precision ax+y
}
int main(void) {
  int N = 1<<20; 
  float *x, *y, *devx, *devy; // host and device pointers
  x = (float*)malloc(N*sizeof(float)); // points to host array
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&devx, N*sizeof(float));  // points to device array
  hipMalloc(&devy, N*sizeof(float));

  for (int i=0; i<N; i++) {
    x[i] = 1.0f; // initialize host arrays
    y[i] = 2.0f;
  }
  // source: host pointer
  // dest: device pointer
  hipMemcpy(devx, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(devy, y, N*sizeof(float), hipMemcpyHostToDevice);
  // launch kernel 
  saxpy<<<(N+255)/256,256>>>(N, 2.0f, devx, devy);

	// check for kernel errors
	// checks for invalid config params
	// sync errors, such as too many threads per block
	hipError_t errSync = hipGetLastError();
	// async errors, such as out of bound memory access
	// needs a device sync, so is expensive..
	// use with care. ie not in production
	hipError_t errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess)
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(hipGetLastError()));

	// cp back results after running kernel function
  hipMemcpy(y, devy, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for(int i=0; i<N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error %.*f\n", 6, maxError);			

  free(x);  // free host mem
  free(y);
  hipFree(devx);  // free device mem
  hipFree(devy);
}
